#include "hip/hip_runtime.h"

#include "complex.h"

extern "C" {

__device__
unsigned npnts = 3;

__device__
double2 pnts[] = {
  {0.0, 0.0},
  {1.0, 0.0},
  {0.5, 0.86602540378}
  //{1.0, 0.0},
  //{0.0, 1.0}
};

__device__
double2 fetch_initial_point(unsigned long i) {
  i = i % npnts;
  return pnts[i];
}

__device__
double2 iterate_point(double2 val, unsigned long i, double2 ipnt, unsigned long func_n) {
  func_n = func_n % npnts;
  return 0.5 * val + 0.5 * pnts[func_n];
}

}

