#include "hip/hip_runtime.h"

#include "complex.h"

extern "C" {

__device__
double2 fetch_initial_point(unsigned long i) {
  i = i % 5;
  switch(i) {
    case 0:
      return (double2){0.0,0.0};
    case 1:
      return (double2){1.0, 0.0};
    case 2:
      return (double2){0.0, 1.0};
    case 3:
      return (double2){1.0, 1.0};
    case 4:
      return (double2){0.5, 0.5};
  }
  return (double2){0.0, 0.0};
}

__device__
double2 iterate_point(double2 val, unsigned long i, double2 ipnt, unsigned long func_n) {
  func_n = func_n % 3;
  switch (func_n) {
    case 0:
      return 0.5 * val;
    case 1:
      return 0.5 * val + (double2){0.5, 0.0};
    case 2:
      return 0.5 * val + (double2){0.0, 0.5};
  }
  return val;
}

}

