

#include <hip/hip_runtime.h>
#include "complex.h"

extern "C" {

__device__
double2 fetch_initial_point(unsigned long i) {
  return (double2){1.0,1.0};
}

__device__
static double _z = 1.0, h = 0.0001, sigma = 10.0, rho = 28.0, beta = 2.6666666667;

__device__
double2 iterate_point(double2 val, unsigned long i, double2 ipnt, unsigned long func_n) {
  double x = val.x, y = val.y, z = _z;
  //double x = _z, y = val.x, z = val.y;
  double nx, ny, nz;
  nx = x + h * sigma * y - h * sigma * x;
  ny = y + h * x * rho - h * x * z - h * y;
  nz = z + h * x * y - h * beta * z;
  _z = nz;
  return (double2){nx, ny};
  //_z = nx;
  //return (double2){ny, nz};
}

}

