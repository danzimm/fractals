#include "hip/hip_runtime.h"

#include <stdint.h>
#include "complex.h"

extern "C" {

__device__
double2 iterate_value(unsigned long i, double mag, double2 val, double2 coord) {
  return complex_pow(val, (double2){8.8, 0.0}) + coord;
}

}

