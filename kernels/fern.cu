

#include <hip/hip_runtime.h>
#include "complex.h"

extern "C" {

__device__
double2 fetch_initial_point(unsigned long i) {
  return (double2){0.0, 0.0};
}

__device__
double2 iterate_point(double2 val, unsigned long i, double2 ipnt, unsigned long func_n) {
  if (func_n < 42949673) {
    func_n = 0;
  } else if (func_n < 3693671875) {
    func_n = 1;
  } else if (func_n < 3994319586) {
    func_n = 2;
  } else {
    func_n = 3;
  }

  switch (func_n) {
    case 0:
      return (double2){0.0, 0.16 * val.y};
    case 1:
      return (double2){0.85 * val.x + 0.04 * val.y, -0.04 * val.x + 0.85 * val.y + 1.6};
    case 2:
      return (double2){0.2 * val.x - 0.26 * val.y, 0.23 * val.x + 0.22 * val.y + 1.6};
    case 3:
      return (double2){-0.15 * val.x + 0.28 * val.y, 0.26 * val.x + 0.24 * val.y + 0.44};
  }
  return val;
}

}

