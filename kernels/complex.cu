
#include <hip/hip_runtime.h>

__device__
const double2 operator-(const double2& vala, const double2& valb) {
  return (double2){vala.x - valb.x, vala.y - valb.y};
}

__device__
const double2 operator+(const double2& vala, const double2& valb) {
  return (double2){vala.x + valb.x, vala.y + valb.y};
}

__device__
const double2 operator*(const double& vala, const double2& valb) {
  return (double2){vala * valb.x, vala * valb.y};
}

__device__
const double2 operator*(const double2& vala, const double& valb) {
  return valb * vala;
}

__device__ double2 complex_exp(double2 val) {
  return exp(val.x) * (double2){cos(val.y), sin(val.y)};
}

__device__ double pown(double val, unsigned long i) {
  unsigned long j;
  double ret = val;
  for (j = 1; j < i; j++) {
    ret = ret * val;
  }
  return ret; 
}

__device__ double complex_mag2(double2 val) {
  return pown(val.x, 2) + pown(val.y, 2);
}

__device__ double complex_mag(double2 val) {
  return sqrt(complex_mag2(val));
}

__device__ double2 complex_ln(double2 val) {
  return (double2){log(complex_mag(val)), atan2(val.y, val.x)};
}

__device__ double2 complex_mult(double2 vala, double2 valb) {
  return (double2){vala.x * valb.x - vala.y * valb.y, vala.x * valb.y + vala.y * valb.x};
}

__device__ double2 complex_pow(double2 val, double2 w) {
  return complex_exp(complex_mult(w, complex_ln(val)));
}

__device__ double2 complex_pown(double2 val, unsigned long n) {
  double2 ret = val;
  unsigned long i;
  for (i = 1; i < n; i++) {
    ret = complex_mult(ret, val);
  }
  return ret;
}

__device__ double2 complex_divide(double2 vala, double2 valb) {
  double diver = 1 / complex_mag2(valb);
  return diver * complex_mult(vala, (double2){valb.x, -valb.y});
}


