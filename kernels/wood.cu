#include "hip/hip_runtime.h"

#include "complex.h"

extern "C" {

__device__ const unsigned npnts = 3;

__device__
double2 pnts[npnts] = {
  {0.0, 0.0},
  {1.0, 0.0},
  {0.0, 0.86602540378}
};

__device__
double2 fetch_initial_point(unsigned long i) {
  i = i % npnts;
  return pnts[i];
}

__device__
double2 iterate_value(double2 val, unsigned long i, double2 ipnt, unsigned long func_n) {
  func_n = func_n % npnts;
  return (double)(func_n+1) / (double)npnts * (val + pnts[func_n]);
}

}

