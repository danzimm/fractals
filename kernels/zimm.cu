#include "hip/hip_runtime.h"

#include <stdint.h>
#include "complex.cu"

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord) {
  double2 val = coord;
  double mag = 0.0f;
  unsigned long i = 0;
  while (i < maxiter) {
    //val = complex_pow(val, (double2){3.7, 0.0}) - complex_pow(val, (double2)(1.5, 0.0)) + coord;
    val = complex_pow(val, (double2){2.1, 0.0}) - complex_pow(val, (double2){1.7, 0.0}) + coord;
    mag = complex_mag2(val);
    if (mag >= escape) {
      break;
    }
    i++;
  }
  *ii = i;
  *magg = mag;
}

}

