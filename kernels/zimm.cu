#include "hip/hip_runtime.h"

#include <stdint.h>
#include "complex.h"

extern "C" {

__device__
double2 iterate_value(unsigned long i, double mag, double2 val, double2 coord) {
  //return complex_pow(val, (double2){3.7, 0.0}) - complex_pow(val, (double2)(1.5, 0.0)) + coord;
  return complex_pow(val, (double2){2.1, 0.0}) - complex_pow(val, (double2){1.7, 0.0}) + coord;
}

}

