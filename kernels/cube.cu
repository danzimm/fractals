#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.h"

extern "C" {

__device__
double2 iterate_value(unsigned long i, double mag, double2 val, double2 coord) {
  return (val - complex_divide(complex_pown(val, 3) - (double2){1.0, 0.0}, 3 * complex_pown(val, 2)));
}

}

