#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.cu"

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord) {
  double2 val = coord;
  double mag = 0.0f;
  unsigned long i = 0;
  while (i < maxiter) {
    val = (val - complex_divide(complex_pown(val, 3) - (double2){1.0, 0.0}, 3 * complex_pown(val, 2)));
    mag = complex_mag2(val);
    if (mag >= escape) {
      break;
    }
    i++;
  }
  *ii = i;
  *magg = mag;
}

}

