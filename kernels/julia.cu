#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.cu"

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord) {
  double2 val = coord;
  double2 c = {-0.4, 0.6};
  double mag = 0.0f;
  unsigned long i = 0;
  while (i < maxiter) {
    val = complex_pown(val, 2) + c;
    mag = complex_mag2(val);
    if (mag >= escape) {
      break;
    }
    i++;
  }
  *ii = i;
  *magg = mag;
}

}

