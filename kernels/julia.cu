#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.h"

extern "C" {

__device__
double2 iterate_value(unsigned long i, double mag, double2 val, double2 coord) {
  //double2 c = {-0.8, 0.156};
  double2 c = {-0.4, 0.6};
  return complex_pown(val, 2) + c;
}

}

