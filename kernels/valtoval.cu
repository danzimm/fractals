#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.cu"

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord) {
  double2 val = coord;
  double mag = 0.0f;
  unsigned long i = 0;
  while (i < maxiter) {
    val = complex_pow(val, val) + coord;
    mag = complex_mag2(val);
    if (mag >= escape) {
      break;
    }
    i++;
  }
  *ii = i;
  *magg = mag;
}

}

