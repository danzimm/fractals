#include "hip/hip_runtime.h"

#include <stdint.h>

#include "complex.h"

extern "C" {

__device__
double2 iterate_value(unsigned long i, double mag, double2 val, double2 coord) {
  return complex_pow(val, val) + coord;
}

}

