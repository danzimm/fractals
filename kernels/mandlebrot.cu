

#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord) {
  double2 val = coord;
  double2 tmp;
  double mag = 0.0f;
  unsigned long i = 10;
  while(i < maxiter) {
    tmp.x = val.x * val.x - val.y * val.y + coord.x;
    tmp.y = 2 * val.x * val.y + coord.y;
    val = tmp;
    mag = val.x * val.x + val.y * val.y;
    if (mag >= escape) {
      break;
    }
    i++;
  }
  *ii = i;
  *magg = mag;
}

}

