#include "hip/hip_runtime.h"

#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include "common.h"
#include "complex.cu"

extern "C" {

__device__
double2 fetch_initial_point(unsigned long i);

__device__
double2 iterate_point(double2 val, unsigned long i, double2 ipnt, unsigned long func_n);

__global__
void initrng(hiprandState *const rng_states, const unsigned int seed) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &rng_states[tid]);
}

__global__
void gencoord(colored_point *points, unsigned long npnts, metadata *meta, hiprandState *const rng_states) {
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = rng_states[tid];
  double2 initial_point, val;
  unsigned long i = 0, ai = 2 * npnts * tid + 2 * i, func_n = 0;
  if (meta->first_run) {
    initial_point = fetch_initial_point(tid);
  } else {
    initial_point = (double2){meta->previous_coords[2 * tid + 0], meta->previous_coords[2 * tid + 1]};
  }
  val = initial_point;
  
  while (i < npnts) {
    func_n = hiprand(&localState);
    points[ai].coord[0] = val.x;
    points[ai].coord[1] = val.y;
    points[ai].color[0] = 0;
    points[ai].color[1] = 255;
    points[ai].color[2] = 0;
    points[ai].color[3] = 255;
    val = iterate_point(val, i, initial_point, func_n);
    i++;
    ai = npnts * tid + i;
  }
  meta->previous_coords[2 * tid + 0] = val.x;
  meta->previous_coords[2 * tid + 1] = val.y;
}

}

