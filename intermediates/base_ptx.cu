#include "hip/hip_runtime.h"

#include <stdint.h>
#include "common.h"

extern "C" {

__device__
void processPixel(unsigned long *ii, double *magg, unsigned long maxiter, double escape, double2 coord);

__device__
void colorizePixel(double pixel[4], double mag, double escape, double i, double maxiter);

__global__
void genimage(uint8_t *pixels, size_t pitch, unsigned long width, unsigned long height, unsigned long yoffset, metadata *meta) {
  unsigned long maxiter = meta->maxiter;
  float4 frame = {meta->frame[0], meta->frame[1], meta->frame[2], meta->frame[3]};
  double escape = meta->escape;
  double color[4] = {meta->color[0], meta->color[1], meta->color[2], meta->color[3]};
  unsigned block_size = blockDim.x;
  uint2 location = {blockIdx.x*block_size, blockIdx.y*block_size + yoffset};
  ulong2 pixel_location = {threadIdx.x, threadIdx.y};
  ulong2 real_location = {location.x + pixel_location.x, location.y + pixel_location.y};
  if (real_location.x >= width || real_location.y >= height)
    return;
  
  double pixel[4] = {color[0], color[1], color[2], color[3]};

  double2 coordsize = {frame.y - frame.x, frame.w - frame.z};
  double2 dimgsize = {(double)width, (double)height};
  double2 dpos = {(double)real_location.x, (double)real_location.y};

  double2 coord = {frame.x + coordsize.x * dpos.x / dimgsize.x, frame.z + coordsize.y * (dimgsize.y-dpos.y) / dimgsize.y};
  unsigned long i = 0;
  double mag = 0.0;
  processPixel(&i, &mag, maxiter, escape, coord);
  
  colorizePixel(pixel, mag, escape, i, maxiter);

  uint8_t *row = (uint8_t *)((char *)pixels + (real_location.y - yoffset) * pitch);
  row[real_location.x * 4 + 0] = (uint8_t)(pixel[0] * 255.0);
  row[real_location.x * 4 + 1] = (uint8_t)(pixel[1] * 255.0);
  row[real_location.x * 4 + 2] = (uint8_t)(pixel[2] * 255.0);
  row[real_location.x * 4 + 3] = (uint8_t)(pixel[3] * 255.0);
}

}

