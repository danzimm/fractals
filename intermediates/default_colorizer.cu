
#include <hip/hip_runtime.h>

extern "C" {

__device__
void colorizePixel(double pixel[4], double mag, double escape, double i, double maxiter) {

  double darkener;
  if (i < maxiter) {
    double inp = (double)i / (double)maxiter;
    double x = escape / mag;
    x = (x / (double)maxiter + inp) - 0.2;
    darkener = 1 / (1.0 + exp(-15.0 * x));
  } else {
    darkener = 0.0f;
  }
  pixel[0] *= darkener;
  pixel[1] *= darkener;
  pixel[2] *= darkener;

}

}

